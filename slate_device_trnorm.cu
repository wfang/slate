#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// Copyright (c) 2017, University of Tennessee
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the University of Tennessee nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL UNIVERSITY OF TENNESSEE BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//------------------------------------------------------------------------------
// This research was supported by the Exascale Computing Project (17-SC-20-SC),
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.
//------------------------------------------------------------------------------
// Need assistance with the SLATE software? Join the "SLATE User" Google group
// by going to https://groups.google.com/a/icl.utk.edu/forum/#!forum/slate-user
// and clicking "Apply to join group". Upon acceptance, email your questions and
// comments to <slate-user@icl.utk.edu>.
//------------------------------------------------------------------------------

#include "slate_device.hh"

#include <cstdio>
#include <hip/hip_complex.h>

namespace slate {
namespace device {

//------------------------------------------------------------------------------
/// max that propogates nan consistently:
///     max_nan( 1,   nan ) = nan
///     max_nan( nan, 1   ) = nan
///
/// For x=nan, y=1:
/// nan < y is false, yields x (nan)
///
/// For x=1, y=nan:
/// x < nan    is false, would yield x, but
/// isnan(nan) is true, yields y (nan)
///
template <typename T>
__host__ __device__
inline T max_nan(T x, T y)
{
    return (isnan(y) || (x) < (y) ? (y) : (x));
}

//------------------------------------------------------------------------------
/// Max reduction of n-element array x, leaving total in x[0]. Propogates NaN
/// values consistently.
/// With k threads, can reduce array up to 2*k in size. Assumes number of
/// threads <= 1024, which is the current max number of CUDA threads.
///
/// @param[in] n
///		Size of array.
///
/// @param[in] tid
///     Thread id.
///
/// @param[in] x
///     Array of dimension n. On exit, x[0] = max(x[0], ..., x[n-1]);
///     the rest of x is overwritten.
///
template <typename T>
__device__ void
max_nan_reduce(int n, int tid, T* x)
{
    if (n > 1024) { if (tid < 1024 && tid + 1024 < n) { x[tid] = max_nan(x[tid], x[tid+1024]); }  __syncthreads(); }
    if (n >  512) { if (tid <  512 && tid +  512 < n) { x[tid] = max_nan(x[tid], x[tid+ 512]); }  __syncthreads(); }
    if (n >  256) { if (tid <  256 && tid +  256 < n) { x[tid] = max_nan(x[tid], x[tid+ 256]); }  __syncthreads(); }
    if (n >  128) { if (tid <  128 && tid +  128 < n) { x[tid] = max_nan(x[tid], x[tid+ 128]); }  __syncthreads(); }
    if (n >   64) { if (tid <   64 && tid +   64 < n) { x[tid] = max_nan(x[tid], x[tid+  64]); }  __syncthreads(); }
    if (n >   32) { if (tid <   32 && tid +   32 < n) { x[tid] = max_nan(x[tid], x[tid+  32]); }  __syncthreads(); }
    if (n >   16) { if (tid <   16 && tid +   16 < n) { x[tid] = max_nan(x[tid], x[tid+  16]); }  __syncthreads(); }
    if (n >    8) { if (tid <    8 && tid +    8 < n) { x[tid] = max_nan(x[tid], x[tid+   8]); }  __syncthreads(); }
    if (n >    4) { if (tid <    4 && tid +    4 < n) { x[tid] = max_nan(x[tid], x[tid+   4]); }  __syncthreads(); }
    if (n >    2) { if (tid <    2 && tid +    2 < n) { x[tid] = max_nan(x[tid], x[tid+   2]); }  __syncthreads(); }
    if (n >    1) { if (tid <    1 && tid +    1 < n) { x[tid] = max_nan(x[tid], x[tid+   1]); }  __syncthreads(); }
}

//------------------------------------------------------------------------------
/// Sum reduction of n-element array x, leaving total in x[0].
/// With k threads, can reduce array up to 2*k in size. Assumes number of
/// threads <= 1024 (which is current max number of CUDA threads).
///
/// @param[in] n
///		Size of array.
///
/// @param[in] tid
///     Thread id.
///
/// @param[in] x
///     Array of dimension n. On exit, x[0] = sum(x[0], ..., x[n-1]);
///     rest of x is overwritten.
///
template <typename T>
__device__ void
sum_reduce(int n, int tid, T* x)
{
    if (n > 1024) { if (tid < 1024 && tid + 1024 < n) { x[tid] += x[tid+1024]; }  __syncthreads(); }
    if (n >  512) { if (tid <  512 && tid +  512 < n) { x[tid] += x[tid+ 512]; }  __syncthreads(); }
    if (n >  256) { if (tid <  256 && tid +  256 < n) { x[tid] += x[tid+ 256]; }  __syncthreads(); }
    if (n >  128) { if (tid <  128 && tid +  128 < n) { x[tid] += x[tid+ 128]; }  __syncthreads(); }
    if (n >   64) { if (tid <   64 && tid +   64 < n) { x[tid] += x[tid+  64]; }  __syncthreads(); }
    if (n >   32) { if (tid <   32 && tid +   32 < n) { x[tid] += x[tid+  32]; }  __syncthreads(); }
    if (n >   16) { if (tid <   16 && tid +   16 < n) { x[tid] += x[tid+  16]; }  __syncthreads(); }
    if (n >    8) { if (tid <    8 && tid +    8 < n) { x[tid] += x[tid+   8]; }  __syncthreads(); }
    if (n >    4) { if (tid <    4 && tid +    4 < n) { x[tid] += x[tid+   4]; }  __syncthreads(); }
    if (n >    2) { if (tid <    2 && tid +    2 < n) { x[tid] += x[tid+   2]; }  __syncthreads(); }
    if (n >    1) { if (tid <    1 && tid +    1 < n) { x[tid] += x[tid+   1]; }  __syncthreads(); }
}

//------------------------------------------------------------------------------
/// Overloaded versions of absolute value on device.
__host__ __device__
inline float abs(float x)
{
    return fabsf(x);
}

__host__ __device__
inline double abs(double x)
{
    return fabs(x);
}

__host__ __device__
inline float abs(hipFloatComplex x)
{
    return hipCabsf(x);
}

__host__ __device__
inline double abs(hipDoubleComplex x)
{
    return hipCabs(x);
}

///-----------------------------------------------------------------------------
/// Square of number.
/// @return x^2
template <typename scalar_t>
__host__ __device__
inline scalar_t sqr(scalar_t x)
{
    return x*x;
}

//------------------------------------------------------------------------------
/// Adds two scaled, sum-of-squares representations.
/// On exit, scale1 and sumsq1 are updated such that:
///     scale1^2 sumsq1 := scale1^2 sumsq1 + scale2^2 sumsq2.
template <typename real_t>
__host__ __device__
void add_sumsq(
    real_t&       scale1, real_t&       sumsq1,
    real_t const& scale2, real_t const& sumsq2 )
{
    if (scale1 > scale2) {
        sumsq1 = sumsq1 + sumsq2*sqr(scale2 / scale1);
        // scale1 stays same
    }
    else {
        sumsq1 = sumsq1*sqr(scale1 / scale2) + sumsq2;
        scale1 = scale2;
    }
}

//------------------------------------------------------------------------------
/// Adds new value to scaled, sum-of-squares representation.
/// On exit, scale and sumsq are updated such that:
///     scale^2 sumsq := scale^2 sumsq + (absx)^2
template <typename real_t>
__host__ __device__
inline void add_sumsq(
    real_t& scale, real_t& sumsq,
    real_t absx)
{
    if (scale < absx) {
        sumsq = 1 + sumsq * sqr(scale / absx);
        scale = absx;
    }
    else {
        sumsq = sumsq + sqr(absx / scale);
    }
}

//------------------------------------------------------------------------------
/// Finds the largest absolute value of elements, for each tile in tiles.
/// Each thread block deals with one tile.
/// Each thread deals with one row, followed by a reduction.
/// Uses dynamic shared memory array of length sizeof(real_t) * m.
/// Kernel assumes non-trivial tiles (m, n >= 1).
/// Launched by trnorm().
///
/// @param[in] m
///     Number of rows of each tile. m >= 1.
///     Also the number of threads per block (blockDim.x), hence,
///     m <= 1024 for current CUDA architectures (2.x to 6.x).
///
/// @param[in] n
///     Number of columns of each tile. n >= 1.
///
/// @param[in] tiles
///     Array of tiles of dimension gridDim.x,
///     where each tiles[k] is an m-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= m.
///
/// @param[out] tiles_maxima
///     Array of dimension gridDim.x.
///     On exit, tiles_maxima[k] = max_{i, j} abs( A^(k)_(i, j) )
///     for tile A^(k).
///
template <typename scalar_t>
__global__ void trnormMaxKernel(
    lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    scalar_t const* const* tiles, int64_t lda,
    blas::real_type<scalar_t>* tiles_maxima)
{
    using real_t = blas::real_type<scalar_t>;
    scalar_t const* tile = tiles[blockIdx.x];
    scalar_t const* row = &tile[threadIdx.x];

    // Each thread finds max of one row.
    // This does coalesced reads of one column at a time in parallel.
    real_t max = 0;
    if (uplo == lapack::Uplo::Lower) {
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < n) // diag
                max = 1;
            for (int64_t j = 0; j < threadIdx.x && j < n; ++j) // strictly lower
                max = max_nan(max, abs(row[j*lda]));
        }
        else {
            for (int64_t j = 0; j <= threadIdx.x && j < n; ++j) // lower
                max = max_nan(max, abs(row[j*lda]));
        }
    }
    else {
        // Loop backwards (n-1 down to i) to maintain coalesced reads.
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < n) // diag
                max = 1;
            for (int64_t j = n-1; j > threadIdx.x; --j) // strictly upper
                max = max_nan(max, abs(row[j*lda]));
        }
        else {
            for (int64_t j = n-1; j >= threadIdx.x; --j) // upper
                max = max_nan(max, abs(row[j*lda]));
        }
    }

    // Save partial results in shared memory.
    extern __shared__ char dynamic_data[];
    real_t* row_max = (real_t*) dynamic_data;
    row_max[threadIdx.x] = max;
    __syncthreads();

    // Reduction to find max of tile.
    max_nan_reduce(blockDim.x, threadIdx.x, row_max);
    if (threadIdx.x == 0) {
        tiles_maxima[blockIdx.x] = row_max[0];
    }
}

//------------------------------------------------------------------------------
/// Sum of absolute values of each column of elements, for each tile in tiles.
/// Each thread block deals with one tile.
/// Each thread deals with one column.
/// Kernel assumes non-trivial tiles (m, n >= 1).
/// Launched by trnorm().
///
/// @param[in] m
///     Number of rows of each tile. m >= 1.
///
/// @param[in] n
///     Number of columns of each tile. n >= 1.
///     Also the number of threads per block (blockDim.x), hence,
///     n <= 1024 for current CUDA architectures (2.x to 6.x).
///
/// @param[in] tiles
///     Array of tiles of dimension gridDim.x,
///     where each tiles[k] is an m-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= m.
///
/// @param[out] tiles_sums
///     Array of dimension gridDim.x * blockDim.x.
///     On exit, tiles_sums[k*n + j] = max_{i} abs( A^(k)_(i, j) )
///     for row j of tile A^(k).
///
template <typename scalar_t>
__global__ void trnormOneKernel(
    lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    scalar_t const* const* tiles, int64_t lda,
    blas::real_type<scalar_t>* tiles_sums)
{
    using real_t = blas::real_type<scalar_t>;
    scalar_t const* tile = tiles[blockIdx.x];
    scalar_t const* column = &tile[lda*threadIdx.x];

    // Each thread sums one column.
    // todo: this doesn't do coalesced reads
    real_t sum = 0;
    if (uplo == lapack::Uplo::Lower) {
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < m) // diag
                sum += 1;
            for (int64_t i = threadIdx.x+1; i < m; ++i) // strictly lower
                sum += abs(column[i]);
        }
        else {
            for (int64_t i = threadIdx.x; i < m; ++i) // lower
                sum += abs(column[i]);
        }
    }
    else {
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < m) // diag
                sum += 1;
            for (int64_t i = 0; i < threadIdx.x && i < m; ++i) // strictly upper
                sum += abs(column[i]);
        }
        else {
            for (int64_t i = 0; i <= threadIdx.x && i < m; ++i) // upper
                sum += abs(column[i]);
        }
    }

    real_t* tile_sums = &tiles_sums[blockIdx.x*n];
    tile_sums[threadIdx.x] = sum;
}

//------------------------------------------------------------------------------
/// Sum of absolute values of each row of elements, for each tile in tiles.
/// Each thread block deals with one tile.
/// Each thread deals with one row.
/// Kernel assumes non-trivial tiles (m, n >= 1).
/// Launched by trnorm().
///
/// @param[in] m
///     Number of rows of each tile. m >= 1.
///     Also the number of threads per block, hence,
///     m <= 1024 for current CUDA architectures (2.x to 6.x).
///
/// @param[in] n
///     Number of columns of each tile. n >= 1.
///
/// @param[in] tiles
///     Array of tiles of dimension gridDim.x,
///     where each tiles[k] is an m-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= m.
///
/// @param[out] tiles_sums
///     Array of dimension gridDim.x * blockDim.x.
///     On exit, tiles_sums[k*m + i] = sum_{j} abs( A^(k)_(i, j) )
///     for row i of tile A^(k).
///
template <typename scalar_t>
__global__ void trnormInfKernel(
    lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    scalar_t const* const* tiles, int64_t lda,
    blas::real_type<scalar_t>* tiles_sums)
{
    using real_t = blas::real_type<scalar_t>;
    scalar_t const* tile = tiles[blockIdx.x];
    scalar_t const* row = &tile[threadIdx.x];

    // Each thread sums one row.
    // This does coalesced reads of one column at a time in parallel.
    real_t sum = 0;
    if (uplo == lapack::Uplo::Lower) {
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < n) // diag
                sum += 1;
            for (int64_t j = 0; j < threadIdx.x && j < n; ++j) // strictly lower
                sum += abs(row[j*lda]);
        }
        else {
            for (int64_t j = 0; j <= threadIdx.x && j < n; ++j) // lower
                sum += abs(row[j*lda]);
        }
    }
    else {
        // Loop backwards (n-1 down to i) to maintain coalesced reads.
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < n) // diag
                sum += 1;
            for (int64_t j = n-1; j > threadIdx.x; --j) // strictly upper
                sum += abs(row[j*lda]);
        }
        else {
            for (int64_t j = n-1; j >= threadIdx.x; --j) // upper
                sum += abs(row[j*lda]);
        }
    }

    real_t* tile_sums = &tiles_sums[blockIdx.x*m];
    tile_sums[threadIdx.x] = sum;
}

//------------------------------------------------------------------------------
/// Sum of squares, in scaled representation, for each tile in tiles.
/// Each thread block deals with one tile.
/// Each thread deals with one row, followed by a reduction.
/// Kernel assumes non-trivial tiles (m, n >= 1).
/// Launched by trnorm().
///
/// @param[in] m
///     Number of rows of each tile. m >= 1.
///
/// @param[in] n
///     Number of columns of each tile. n >= 1.
///     Also the number of threads per block, hence,
///     n <= 1024 for current CUDA architectures (2.x to 6.x).
///
/// @param[in] tiles
///     Array of tiles of dimension blockDim.x,
///     where each tiles[k] is an m-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= m.
///
/// @param[out] tiles_values
///     Array of dimension 2 * blockDim.x.
///     On exit,
///         tiles_values[2*k + 0] = scale
///         tiles_values[2*k + 1] = sumsq
///     such that scale^2 * sumsq = sum_{i,j} abs( A^(k)_{i,j} )^2
///     for tile A^(k).
///
template <typename scalar_t>
__global__ void trnormFroKernel(
    lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    scalar_t const* const* tiles, int64_t lda,
    blas::real_type<scalar_t>* tiles_values)
{
    using real_t = blas::real_type<scalar_t>;
    scalar_t const* tile = tiles[blockIdx.x];
    scalar_t const* row = &tile[threadIdx.x];

    // Each thread finds sum-of-squares of one row.
    // This does coalesced reads of one column at a time in parallel.
    real_t scale = 0;
    real_t sumsq = 1;
    if (uplo == lapack::Uplo::Lower) {
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < n) // diag
                add_sumsq(scale, sumsq, real_t(1));
            for (int64_t j = 0; j < threadIdx.x && j < n; ++j) // strictly lower
                add_sumsq(scale, sumsq, abs(row[j*lda]));
        }
        else {
            for (int64_t j = 0; j <= threadIdx.x && j < n; ++j) // lower
                add_sumsq(scale, sumsq, abs(row[j*lda]));
        }
    }
    else {
        // Loop backwards (n-1 down to i) to maintain coalesced reads.
        if (diag == lapack::Diag::Unit) {
            if (threadIdx.x < n) // diag
                add_sumsq(scale, sumsq, real_t(1));
            for (int64_t j = n-1; j > threadIdx.x; --j) // strictly upper
                add_sumsq(scale, sumsq, abs(row[j*lda]));
        }
        else {
            for (int64_t j = n-1; j >= threadIdx.x; --j) // upper
                add_sumsq(scale, sumsq, abs(row[j*lda]));
        }
    }

    // Save partial results in shared memory.
    extern __shared__ char dynamic_data[];
    real_t* row_scale = (real_t*) &dynamic_data[0];
    real_t* row_sumsq = &row_scale[m];
    row_scale[threadIdx.x] = scale;
    row_sumsq[threadIdx.x] = sumsq;
    __syncthreads();

    // Reduction to find sum-of-squares of tile.
    // todo: parallel reduction.
    if (threadIdx.x == 0) {
        real_t tile_scale = row_scale[0];
        real_t tile_sumsq = row_sumsq[0];
        for (int64_t i = 1; i < m; ++i)
            add_sumsq(tile_scale, tile_sumsq, row_scale[i], row_sumsq[i]);

        tiles_values[blockIdx.x*2 + 0] = tile_scale;
        tiles_values[blockIdx.x*2 + 1] = tile_sumsq;
    }
}

//------------------------------------------------------------------------------
/// Batched routine that returns the largest absolute value of elements for
/// each tile in Aarray. Sets
///     tiles_maxima[k] = max_{i, j}( abs( A^(k)_(i, j) )),
/// for each tile A^(k), where
/// A^(k) = Aarray[k],
/// k = 0, ..., blockDim.x-1,
/// i = 0, ..., m-1,
/// j = 0, ..., n-1.
///
/// @param[in] m
///     Number of rows of each tile. m >= 0.
///
/// @param[in] n
///     Number of columns of each tile. n >= 0.
///     Currently, n <= 1024 due to CUDA implementation.
///
/// @param[in] Aarray
///     Array in GPU memory of dimension batch_count, containing pointers to tiles,
///     where each Aarray[k] is an m-by-n matrix stored in an lda-by-n array in GPU memory.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= m.
///
/// @param[out] values
///     Array in GPU memory.
///     - Norm::Max: dimension batch_count.
///         On exit, values[k] = max_{i, j} abs( A^(k)_(i, j) )
///         for 0 <= k < batch_count.
///
///     - Norm::One: dimension batch_count * n.
///         On exit, values[k*n + j] = sum_{i} abs( A^(k)_(i, j) )
///         for 0 <= k < batch_count, 0 <= j < n.
///
///     - Norm::Inf: dimension batch_count * m.
///         On exit, values[k*m + i] = sum_{j} abs( A^(k)_(i, j) )
///         for 0 <= k < batch_count, 0 <= i < m.
///
///     - Norm::Max: dimension batch_count * 2.
///         On exit,
///             values[k*2 + 0] = scale_k
///             values[k*2 + 1] = sumsq_k
///         where scale_k^2 sumsq_k = sum_{i,j} abs( A^(k)_(i, j) )^2
///         for 0 <= k < batch_count.
///
/// @param[in] batch_count
///     Size of Aarray. batch_count >= 0.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t>
void trnorm(
    lapack::Norm norm, lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    scalar_t const* const* Aarray, int64_t lda,
    blas::real_type<scalar_t>* values, int64_t batch_count,
    hipStream_t stream)
{
    using real_t = blas::real_type<scalar_t>;

    // quick return
    if (batch_count == 0)
        return;

    //---------
    // max norm
    if (norm == lapack::Norm::Max) {
        if (m == 0 || n == 0) {
            hipMemsetAsync(values, 0, sizeof(real_t) * batch_count, stream);
        }
        else {
            assert(m <= 1024);
            // Max 1024 threads * 8 bytes = 8 KiB shared memory in double [complex].
            trnormMaxKernel<<<batch_count, m, sizeof(real_t) * m, stream>>>
                (uplo, diag, m, n, Aarray, lda, values);
        }
    }
    //---------
    // one norm
    else if (norm == lapack::Norm::One) {
        if (m == 0 || n == 0) {
            hipMemsetAsync(values, 0, sizeof(real_t) * batch_count * n, stream);
        }
        else {
            assert(n <= 1024);
            trnormOneKernel<<<batch_count, n, 0, stream>>>
                (uplo, diag, m, n, Aarray, lda, values);
        }
    }
    //---------
    // inf norm
    else if (norm == lapack::Norm::Inf) {
        if (m == 0 || n == 0) {
            hipMemsetAsync(values, 0, sizeof(real_t) * batch_count * m, stream);
        }
        else {
            assert(m <= 1024);
            trnormInfKernel<<<batch_count, m, 0, stream>>>
                (uplo, diag, m, n, Aarray, lda, values);
        }
    }
    //---------
    // Frobenius norm
    else if (norm == lapack::Norm::Fro) {
        if (m == 0 || n == 0) {
            hipMemsetAsync(values, 0, sizeof(real_t) * batch_count * 2, stream);
        }
        else {
            assert(m <= 1024);
            // Max 1024 threads * 16 bytes = 16 KiB shared memory in double [complex].
            trnormFroKernel<<<batch_count, m, sizeof(real_t) * m * 2, stream>>>
                (uplo, diag, m, n, Aarray, lda, values);
        }
    }

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
// Explicit instantiations.
template
void trnorm(
    lapack::Norm norm, lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    float const* const* Aarray, int64_t lda,
    float* values, int64_t batch_count,
    hipStream_t stream);

template
void trnorm(
    lapack::Norm norm, lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    double const* const* Aarray, int64_t lda,
    double* values, int64_t batch_count,
    hipStream_t stream);

template
void trnorm(
    lapack::Norm norm, lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    hipFloatComplex const* const* Aarray, int64_t lda,
    float* values, int64_t batch_count,
    hipStream_t stream);

template
void trnorm(
    lapack::Norm norm, lapack::Uplo uplo, lapack::Diag diag,
    int64_t m, int64_t n,
    hipDoubleComplex const* const* Aarray, int64_t lda,
    double* values, int64_t batch_count,
    hipStream_t stream);

} // namespace device
} // namespace slate
